#include "hip/hip_runtime.h"
/**
 * @file   gpu_cg.h
 * @author Alberto Franco
 * @date   19/05/2010
 * 
 * Contiene i prototipi delle funzioni che vanno ad 
 * agire sulla GPU per il CG e le strutture per agire con tali 
 * funzioni.
 */
#include "gpu_cg.h"

__global__ void applyVector(const int*    row_offset, const int*    column_index,
                            const double* values,     const double* vector, 
                            double*       result,     int           length);


/**
 * Applica il vettore alla matrice e salva i risultati in result.
 * @param row_offset L'offset delle righe della matrice.
 * @param column_index Le posizioni dei non nulli della matrice 
 * @param values I valori della matrice a cui applicare il vettore
 * @param vector Il vettore da applicare.
 * @param result Il vettore in cui salvare i risultati
 * @param length La lunghezza del vettore da applicare.
 */
void gpuApplyVector(const int*    row_offset, const int*    column_index,
                    const double* values,     const double* vector, 
                    double*       result,     int           length)
{
   // Calcolo il numero di blocchi da invocare per eseguire correnttamente
   // l'operazione richiesta.
   int blockCount = length / g_BlockSize;
   
   // Se non � divisibile per la dimensione di blocco
   if(length % g_BlockSize != 0)
   {
      // Incremento il numero di blocchi di uno
      blockCount += 1;
   }
   
   // Invoco il kernel
   applyVector<<<blockCount, g_BlockSize>>>(row_offset, column_index, values,
                                            vector, result, length);
   
}


// Kernel per la divisione degli scalari
__global__ void divide(double* a, double* b)
{
   // Divido i due numeri e metto il risultato in b
   *b = *a / *b;
}

/**
 * Divide i valori e li inserisce nel secondo. a / b
 * @param first Il primo valore (a)
 * @param second Il secondo valore (b) <- a / b
 */
void gpuDivide(double* first, double* second)
{
   // Invoco il kernel
   divide<<<1, 1>>>(first, second);
}



// Kernel per la applicazione matrice vettore
__global__ void applyVector(const int*    row_offset, const int*    column_index,
                            const double* values,     const double* vector, 
                            double*       result,     int           length)
{
   // Calcolo l'id del thread
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   
   // Se sono uscito dalla lunghezza non eseguo operazioni per evitare 
   // di scrivere su memoria non allocata.
   if(idx < length)
   {
      // Inizializzo la cella risultato a zero.
      result[idx] = 0.0;
      
      // Scorro tutti gli elementi della riga che mi interessano e li sommo
      for(int i = row_offset[idx]; i < row_offset[idx +1]; i++)
      {  
         // Aggiungo al risultato il valore della moltiplicazione,
         // i valori che sono nulli nella riga non mi interessano perch�
         // cmq la moltiplicazione � zero.
         result[idx] += values[i] * vector[column_index[i]];
      }
   }
}


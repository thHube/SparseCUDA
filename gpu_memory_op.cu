/**
 * @file   gpu_memory_op.cu
 * @author Alberto Franco
 * @date   19/05/2010
 * 
 * Contiene l'implementazione delle funzioni di gestione della memoria per il
 * metodo del gradiente coniugato.
 */
#include "gpu_cg.h"

/**
 * Alloca sulla scheda grafica un segmento di memoria lungo quanto richiesto.
 * @param size La dimensione in byte del segmento.
 * @return Il puntatore alla zona di memoria.
 */
void* gpuAllocate(int size)
{
   // Uso un puntatore temporaneo
   void* memory_pointer;
   
   // Richiedo l'allocazione di memoria
   hipMalloc(&memory_pointer, size);
   
   // Ritorno la nuova area di memoria allocata.
   return memory_pointer;
}

/**
 * Libera la memoria allocata puntata dal parametro.
 * @param memoryPointer il puntatore alla zona di memoria da liberare
 */
void gpuDelete(void* memoryPointer)
{
   // Libero la zona di memoria puntata
   hipFree(memoryPointer);
}


/**
 * Copia la memoria sull'host e ritorna il puntatore alla zona 
 * di memoria allocata.
 * @param gpuPointer Il punto da cui copiare la memoria.
 * @param size La dimensione in byte della memoria da copiare.
 * @return Il puntatore alla zona di memoria con i dati copiati in RAM.
 */
void* gpuCopyMemoryToHost(void* gpuPointer, int size)
{
   // Mi serve un puntatore temporaneo
   void* ram_pointer = malloc(size);
   
   // Copio la memoria sull'area appena allocata
   hipMemcpy(ram_pointer, gpuPointer, size, hipMemcpyDeviceToHost);
   
   // Ritorno il puntatore alla memoria RAM.
   return ram_pointer;
}


/**
 * Trasferisce la memoria della RAM alla VRAM.
 * @param ramPointer La sorgente da cui copiare la memoria
 * @param gpuPointer La destinazione su cui copiare la memoria
 * @param size La dimensione in byte della memoria da copiare.
 */
void gpuCopyMemoryToDevice(void* ramPointer, void* gpuPointer, int size)
{
   // Invoco la funzione di copia
   hipMemcpy(gpuPointer, ramPointer, size, hipMemcpyHostToDevice);

}

/**
 * Copia memoria dalla scheda alla scheda.
 * @param origin Il puntatore sorgente da cui copiare la memoria.
 * @param destination Il puntatore dove copiare la memoria.
 * @param size La dimensione in byte della memoria da copiare.
 */
void gpuCopyMemory(void* origin, void* destination, int size)
{
   // Invoco la funzione di copia
   hipMemcpy(destination, origin, size, hipMemcpyDeviceToDevice);
}


